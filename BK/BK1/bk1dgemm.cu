#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <timer.h>

__global__ void bk1weighting(int N, double *GWJ, double *res){

  int n = threadIdx.x + blockIdx.x*blockDim.x;

  if(n<N){
    res[n] *= GWJ[n];
  }

}

// to compile: 
//   nvcc -I.  -o hipblasDgemm hipblasDgemm.cu -lcublas
//./hipblasDgemm  168 56  162000
int main(int argc, char **argv) {

  if(argc<4){
    printf("usage: ./hipblasDgemm NrowsA NcolsA NcolsB\n");
    exit(-1);
  }

  // Allocate 3 arrays on CPU
  unsigned long long int NrowsA = atoi(argv[1]);
  unsigned long long int NcolsA = atoi(argv[2]);
  unsigned long long int NcolsB = atoi(argv[3]);
  unsigned long long int NrowsB = NcolsA;
  unsigned long long int NrowsC = NrowsA;
  unsigned long long int NcolsC = NcolsB;
  unsigned long long int NrowsD = NrowsA;
  unsigned long long int NcolsD = NcolsB;
    
  
  double *h_A = (double *)malloc(NrowsA * NcolsA * sizeof(double));
  double *h_B = (double *)malloc(NrowsB * NcolsB * sizeof(double));
  double *h_C = (double *)malloc(NrowsC * NcolsC * sizeof(double));
  double *h_D = (double *)malloc(NrowsD * NcolsD * sizeof(double));
  
  for(int row=0;row<NrowsA;++row){
    for(int col=0;col<NcolsA;++col){
      h_A[row + col*NrowsA] = drand48();
    }
  }
  
  for(int row=0;row<NrowsB;++row){
    for(int col=0;col<NcolsB;++col){
      h_B[row + col*NrowsB] = drand48();
    }
  }
  
  // Allocate 3 arrays on GPU
  double *c_A, *c_B, *c_C, *c_D, *c_WJ;
  hipMalloc(&c_A,NrowsA * NcolsA * sizeof(double));
  hipMalloc(&c_B,NrowsB * NcolsB * sizeof(double));
  hipMalloc(&c_C,NrowsC * NcolsC * sizeof(double));
  hipMalloc(&c_WJ,NrowsC * NcolsC * sizeof(double));
  hipMalloc(&c_D,NrowsD * NcolsD * sizeof(double));
  
  // ....
  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasSetMatrix(NrowsA,NcolsA, sizeof(double), h_A, NrowsA, c_A, NrowsA);
  hipblasSetMatrix(NrowsB,NcolsB, sizeof(double), h_B, NrowsB, c_B, NrowsB);
  hipblasSetMatrix(NrowsC,NcolsC, sizeof(double), h_C, NrowsC, c_C, NrowsC);
  hipblasSetMatrix(NrowsD,NcolsD, sizeof(double), h_D, NrowsD, c_D, NrowsD);
  
  const double alpha = 1.0, beta = 0.0;
  
  // C = beta*C + alpha*A*B
  const unsigned long long int lda = NrowsA, ldb = NrowsB, ldc = NrowsC, ldd = NrowsD;

  int Nmult = 5;
  int Ntests = 10;
  for(int m=0;m<Nmult;++m){

    hipDeviceSynchronize();

    static uint64_t tic = ns();

    hipEvent_t etic, etoc;
    hipEventCreate(&etic);
    hipEventCreate(&etoc);

    hipEventRecord(etic);

    hipblasStatus_t err;
    for(int t=0;t<Ntests;++t){
      
      err = hipblasDgemm(handle,
                        HIPBLAS_OP_N,
                        HIPBLAS_OP_N,
                        NrowsA, NcolsB, NcolsA,
                        &alpha,
                        c_A, lda,
                        c_B, ldb,
                        &beta,
                        c_C, ldc);

      int N = NrowsC*NcolsC;
      int T = 256;
      int B = (N+T-1)/T;

      bk1weighting <<< B, T >>> (N, c_WJ, c_C);

      err = hipblasDgemm(handle,
                        HIPBLAS_OP_T,
                        HIPBLAS_OP_N,
                        NrowsA, NcolsB, NcolsA,
                        &alpha,
                        c_A, lda,
                        c_C, ldc,
                        &beta,
                        c_D, ldd);

      
    }
    
    hipEventRecord(etoc);
    
    if(err != HIPBLAS_STATUS_SUCCESS){
      printf("hipblasDgemm failed, exiting\n");
      exit(-1);
    }

    hipDeviceSynchronize();
    
    static uint64_t toc = ns();
    if(m==Nmult-1){
      
      double elapsed = ((toc-tic)/1.e9)/Ntests;

      float eventElapsed = 0;
      hipEventElapsedTime(&eventElapsed, etic, etoc);
      eventElapsed /= (Ntests*1000.);
      
      double gflop = 2.*NrowsA*NcolsA*NcolsB*2./1.e9;
      
      
      printf("%lld, %lld, %lld, %g, %g, %g, %g %%%% N, elapsed, eventElapsed, gflops, event gflops\n",
             NrowsA, NcolsA, NcolsB, elapsed, eventElapsed, gflop/elapsed, gflop/eventElapsed);
    }
    
  }
  
  hipMemcpy(h_C, c_C, NrowsC*NcolsC*sizeof(double), hipMemcpyDeviceToHost);

#if 0
  for(int n=0;n<100;++n){
    printf("%f ", h_C[n]);
  }
#endif
  
  // TIDY UP
  // Destroy the handle
  hipblasDestroy(handle);
  
  //Free GPU memory
  hipFree(c_A);
  hipFree(c_B);
  hipFree(c_C);
  
  // Free CPU memory
  free(h_A);
  free(h_B);
  free(h_C);
  
  return 0;
}
